#include "hip/hip_runtime.h"
#include "voxelizer.cuh"
#include <string>
#include <iostream>

__global__ void vecSubDArr(int* firX, int* firY, int* firZ,
	int* secX, int* secY, int* secZ,
	int* outX, int* outY, int* outZ) {
	outX[blockIdx.x] = firX[blockIdx.x] - secX[blockIdx.x];
	outY[blockIdx.x] = firY[blockIdx.x] - secY[blockIdx.x];
	outZ[blockIdx.x] = firZ[blockIdx.x] - secZ[blockIdx.x];
	
}

__global__ void vecSubPoint(int* QY, int* QZ,
	int* VX, int* VY, int* VZ,
	int* outX, int* outY, int* outZ) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	outX[index] = blockIdx.x - VX[threadIdx.x];
	outY[index] = *QY - VY[threadIdx.x];
	outZ[index] = *QZ - VZ[threadIdx.x];

	//if (blockIdx.x == 0 && threadIdx.x == 0) printf("x: %d tri: %d QVoutX: %d QVoutY: %d QVoutZ: %d  \n", blockIdx.x, threadIdx.x, outX[index], outY[index], outZ[index]);
}

__global__ void vecSubPointOri(int* PY, int* PZ,
	int* VX, int* VY, int* VZ,
	int* outX, int* outY, int* outZ) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	outX[index] = VX[threadIdx.x] - blockIdx.x;
	outY[index] = VY[threadIdx.x] - *PY;
	outZ[index] = VZ[threadIdx.x] - *PZ;
}

__global__ void vecCross(int* firX, int* firY, int* firZ,
	int* secX, int* secY, int* secZ,
	int* outX, int* outY, int* outZ) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	outX[index] = (firY[threadIdx.x] * secZ[index]) - (firZ[threadIdx.x] * secY[index]);
	outY[index] = (firZ[threadIdx.x] * secX[index]) - (firX[threadIdx.x] * secZ[index]);
	outZ[index] = (firX[threadIdx.x] * secY[index]) - (firY[threadIdx.x] * secX[index]);
	//if (blockIdx.x == 0 && threadIdx.x == 0) printf("firX: %d firY: %d firZ: %d outX: %d outY: %d outZ: %d \n", firX[threadIdx.x], firY[threadIdx.x], firZ[threadIdx.x], outX[index], outY[index], outZ[index]);
}


__global__ void normDotDouble(int* firX, int* firY, int*firZ,
	int* secX, int* secY, int* secZ, int* out){
	int sum = firX[blockIdx.x] * secX[blockIdx.x];
	sum += (firY[blockIdx.x] * secY[blockIdx.x]);
	sum += (firZ[blockIdx.x] * secZ[blockIdx.x]);
	out[blockIdx.x] = sum;
}

__global__ void normDot(int* firX, int* firY, int* firZ,
	int* secX, int* secY, int* secZ,
	bool* check) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int sum = firX[index] * secX[threadIdx.x];
	sum += firY[index] * secY[threadIdx.x];
	sum += firZ[index] * secZ[threadIdx.x];
	if (sum >= 0) check[index] = true;
	
		//if (blockIdx.x == 0) printf("x: %d tri: %d crossX: %d crossY: %d crossZ: %d \n", blockIdx.x, threadIdx.x, firX[index], firY[index], firZ[index]);
		//if (blockIdx.x == 0 && threadIdx.x == 0) printf("x: %d tri: %d normX: %d normY: %d normZ: %d \n", blockIdx.x, threadIdx.x, secX[threadIdx.x], secY[threadIdx.x], secZ[threadIdx.x]);
		if(blockIdx.x == 0 && threadIdx.x == 0) printf("x: %d tri: %d dot prod: %d \n", blockIdx.x, threadIdx.x, sum);
	
}

__global__ void DCALC(int* uu, int* uv, int* vv, int* D){
	D[blockIdx.x] = (uv[blockIdx.x] * uv[blockIdx.x]) - (uu[blockIdx.x] * vv[blockIdx.x]);
}

__global__ void normDotW(int* wx, int* wy, int* wz, int* vecx, int* vecy, int* vecz, int* out){
	int indexW = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = wx[indexW] * vecx[threadIdx.x];
	sum += (wy[indexW] * vecy[threadIdx.x]);
	sum += (wz[indexW] * vecz[threadIdx.x]);
	out[indexW] = sum;
}

__global__ void paramTest(int* uu, int* uv, int* vv, int* wu, int* wv, int* D, bool* intersects) {
	float s, t;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	s = (float)(((uv[threadIdx.x] * wv[index]) - (vv[threadIdx.x] * wu[index]))) / D[threadIdx.x];
	t = (float)(((uv[threadIdx.x] * wu[index]) - (uu[threadIdx.x] * wv[index]))) / D[threadIdx.x];
	if (!(s < 0.0 || s > 1.0) && !(t < 0.0 || (s + t) > 1.0)) intersects[index] = true;

	printf("D: %d X: %d s: %f t: %f\n",D[threadIdx.x], blockIdx.x, s, t);

}

__global__ void checkSum(bool* c1, bool* c2, bool* c3,
	int* actTris, bool* fills) {
	bool checker = false;
	int index;
	int offset = blockIdx.x * (*actTris);
	for (int i = 0; i < (*actTris); i++) {
		index = offset + i;
		checker = (c1[index] && c2[index]) && c3[index];
		if (checker) fills[blockIdx.x] = true;
	}
}


__global__ void angleFind(int* firX, int* firY, int* firZ,
	int* secX, int* secY, int* secZ, float* out) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	float top = firX[index] * secX[index];
	top += firY[index] * secY[index];
	top += firZ[index] * secZ[index];

	float normFir = sqrtf((float)(firX[index] * firX[index] + firY[index] * firY[index] + firZ[index] * firZ[index]));
	float normSec = sqrtf((float)(secX[index] * secX[index] + secY[index] * secY[index] + secZ[index] * secZ[index]));

	float input = top / (normFir * normSec);
	out[index] = acosf(input);
}


__global__ void angleSum(float* ang1, float* ang2, float* ang3, bool* planeInt,
	bool* intersect) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = abs(ang1[index]) + abs(ang2[index]) + abs(ang3[index]);
	//printf("X: %d sum: %f \n", blockIdx.x, sum);
	if (abs(sum - 6.28) < 0.02 && planeInt[index]) intersect[index] = true;
}

__global__ void intersectCount(int* numTris, bool* intersects, bool* outIntersect) {
	int indexOffset = blockIdx.x * (*numTris);
	bool out = false;
	int intersectCount = 0;
	for (int i = 0; i < *numTris; i++) {
		
		if (intersects[indexOffset + i]) { out = true; intersectCount++; }
	}
	outIntersect[blockIdx.x] = out;
	printf("X: %d Intersections: %d \n", blockIdx.x, intersectCount);
}

__global__ void planeIntersect(int* A, int* B, int* C, int* D, int* inY, int* inZ, bool* out) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	int sum = blockIdx.x * A[threadIdx.x];
	sum += (*inY) * B[threadIdx.x];
	sum += (*inZ) * C[threadIdx.x];
	if (abs(sum - D[threadIdx.x]) < 0.1) out[index] = true;
	if ((*inY) == 9 && (*inZ) == 9) {
		printf("Norm: A: %d B: %d C:%d D: %d X: %d \n", A[threadIdx.x], B[threadIdx.x], C[threadIdx.x], D[threadIdx.x], blockIdx.x);
	}
}

void actTriFind(std::vector<int> minZTris,
	std::vector<int> maxZTris,
	std::vector<int> minYTris,
	std::vector<int> maxYTris,
	std::vector<int>& actTris,
	const int y, const int z) {
	int len = minZTris.size();
	bool checkZ;
	bool checkY;
	for (int i = 0; i < len; i++) {
		checkY = (minYTris[i] <= y && maxYTris[i] >= y);
		checkZ = (minZTris[i] <= z && maxZTris[i] >= z);
		if (checkY && checkZ) actTris.push_back(i);
	}
}

namespace voxel {
	void voxelize(
		const std::vector<std::vector<int>> triVecs,
		const std::vector<std::vector<int>> norms,
		const int width, const int height, const int depth,
		const std::vector<int> minZTris,
		const std::vector<int> maxZTris,
		const std::vector<int> minYTris,
		const std::vector<int> maxYTris,
		bool*** fills
	) {
		/*fills = new bool** [depth];
		for (int d = 0; d < depth; d++) {
			fills[d] = new bool* [height];
			for (int h = 0; h < height; h++) {
				fills[d][h] = new bool[width];
			}
		}*/
		int numTris = minZTris.size();
		int size;
		int* ax, * ay, * az,
			* bx, * by, * bz,
			* cx, * cy, * cz;

		int* NX, * NY, * NZ;

		int* d_ax, * d_ay, * d_az,
			* d_bx, * d_by, * d_bz,
			* d_cx, * d_cy, * d_cz;

		int* d_BAX, * d_BAY, * d_BAZ,
			* d_CBX, * d_CBY, * d_CBZ,
			* d_ACX, * d_ACY, * d_ACZ;

		int* d_QAX, * d_QAY, * d_QAZ,
			* d_QBX, * d_QBY, * d_QBZ,
			* d_QCX, * d_QCY, * d_QCZ;

		int* d_QY, * d_QZ;

		int* d_BAQAX, * d_BAQAY, * d_BAQAZ,
			* d_CBQBX, * d_CBQBY, * d_CBQBZ,
			* d_ACQCX, * d_ACQCY, * d_ACQCZ;

		int* d_NX, * d_NY, * d_NZ;
		bool* d_C1, * d_C2, * d_C3;
		bool* d_fill; int* d_numTris;
		std::vector<int> activeTris;
		for (int d = 0; d < depth; d++) {
			for (int h = 0; h < height; h++) {
				//if (h == 9 && d == 9) {
					activeTris.clear();
					actTriFind(minZTris, maxZTris, minYTris, maxYTris, activeTris, h, d);
					if (activeTris.size() == 0) {
						for (int w = 0; w < width; w++) {
							fills[d][h][w] = false;
						}
					}
					else {
						printf("Level: %d height: %d Active tris: %d\n", d, h, activeTris.size());
						int numTris = activeTris.size();
						size = sizeof(int) * numTris;
						ax = (int*)malloc(size); ay = (int*)malloc(size); az = (int*)malloc(size);
						bx = (int*)malloc(size); by = (int*)malloc(size); bz = (int*)malloc(size);
						cx = (int*)malloc(size); cy = (int*)malloc(size); cz = (int*)malloc(size);
						NX = (int*)malloc(size); NY = (int*)malloc(size); NZ = (int*)malloc(size);
						for (int i = 0; i < numTris; i++) {
							std::vector<int> actVecA = triVecs[(activeTris[i] * 3)];
							std::vector<int> actVecB = triVecs[(activeTris[i] * 3) + 1];
							std::vector<int> actVecC = triVecs[(activeTris[i] * 3) + 2];
							ax[i] = actVecA[0]; ay[i] = actVecA[1]; az[i] = actVecA[2];
							bx[i] = actVecB[0]; by[i] = actVecB[1]; bz[i] = actVecB[2];
							cx[i] = actVecC[0]; cy[i] = actVecC[1]; cz[i] = actVecC[2];
							std::vector<int> norm = norms[activeTris[i]];
							NX[i] = norm[0]; NY[i] = norm[1]; NZ[i] = norm[2];
							printf("Tri: %d NormX: %d NormY: %d NormZ: %d\n", i, NX[i], NY[i], NZ[i]);
							if (i == 0) {
								printf("Tri: %d ax: %d ay: %d az: %d \n", i, ax[i], ay[i], az[i]);
								printf("Tri: %d bx: %d by: %d bz: %d \n", i, bx[i], by[i], bz[i]);
								printf("Tri: %d cx: %d cy: %d cz: %d \n", i, cx[i], cy[i], cz[i]);
							}
						}
						hipMalloc((void**)&d_ax, size); hipMalloc((void**)&d_ay, size); hipMalloc((void**)&d_az, size);
						hipMalloc((void**)&d_bx, size); hipMalloc((void**)&d_by, size); hipMalloc((void**)&d_bz, size);
						hipMalloc((void**)&d_cx, size); hipMalloc((void**)&d_cy, size); hipMalloc((void**)&d_cz, size);

						hipMalloc((void**)&d_BAX, size); hipMalloc((void**)&d_BAY, size); hipMalloc((void**)&d_BAZ, size);
						hipMalloc((void**)&d_CBX, size); hipMalloc((void**)&d_CBY, size); hipMalloc((void**)&d_CBZ, size);
						hipMalloc((void**)&d_ACX, size); hipMalloc((void**)&d_ACY, size); hipMalloc((void**)&d_ACZ, size);

						hipMemcpy(d_ax, ax, size, hipMemcpyHostToDevice); hipMemcpy(d_ay, ay, size, hipMemcpyHostToDevice); hipMemcpy(d_az, az, size, hipMemcpyHostToDevice);
						hipMemcpy(d_bx, bx, size, hipMemcpyHostToDevice); hipMemcpy(d_by, by, size, hipMemcpyHostToDevice); hipMemcpy(d_bz, bz, size, hipMemcpyHostToDevice);
						hipMemcpy(d_cx, cx, size, hipMemcpyHostToDevice); hipMemcpy(d_cy, cy, size, hipMemcpyHostToDevice); hipMemcpy(d_cz, cz, size, hipMemcpyHostToDevice);

						free(ax); free(ay); free(az);
						free(bx); free(by); free(bz);
						free(cx); free(cy); free(cz);

						vecSubDArr << <numTris, 1 >> > (d_bx, d_by, d_bz, d_ax, d_ay, d_az, d_BAX, d_BAY, d_BAZ);
						vecSubDArr << <numTris, 1 >> > (d_cx, d_cy, d_cz, d_bx, d_by, d_bz, d_CBX, d_CBY, d_CBZ);
						vecSubDArr << <numTris, 1 >> > (d_ax, d_ay, d_az, d_cx, d_cy, d_cz, d_ACX, d_ACY, d_ACZ);

						hipDeviceSynchronize();

						int rowActTriSize = sizeof(int) * numTris * width;

						hipMalloc((void**)&d_QAX, rowActTriSize); hipMalloc((void**)&d_QAY, rowActTriSize); hipMalloc((void**)&d_QAZ, rowActTriSize);
						hipMalloc((void**)&d_QBX, rowActTriSize); hipMalloc((void**)&d_QBY, rowActTriSize); hipMalloc((void**)&d_QBZ, rowActTriSize);
						hipMalloc((void**)&d_QCX, rowActTriSize); hipMalloc((void**)&d_QCY, rowActTriSize); hipMalloc((void**)&d_QCZ, rowActTriSize);

						int dvarsize = sizeof(int);
						hipMalloc((void**)&d_QY, dvarsize); hipMalloc((void**)&d_QZ, dvarsize);
						hipMemcpy(d_QY, &h, dvarsize, hipMemcpyHostToDevice); hipMemcpy(d_QZ, &d, dvarsize, hipMemcpyHostToDevice);

						vecSubPoint << <width, numTris >> > (d_QY, d_QZ, d_ax, d_ay, d_az, d_QAX, d_QAY, d_QAZ);
						hipDeviceSynchronize();

						vecSubPoint << <width, numTris >> > (d_QY, d_QZ, d_bx, d_by, d_bz, d_QBX, d_QBY, d_QBZ);
						hipDeviceSynchronize();
						vecSubPoint << <width, numTris >> > (d_QY, d_QZ, d_cx, d_cy, d_cz, d_QCX, d_QCY, d_QCZ);

						hipDeviceSynchronize();

						hipFree(d_QY); hipFree(d_QZ);

						hipFree(d_ax); hipFree(d_ay); hipFree(d_az);
						hipFree(d_bx); hipFree(d_by); hipFree(d_bz);
						hipFree(d_cx); hipFree(d_cy); hipFree(d_cz);

						hipMalloc((void**)&d_BAQAX, rowActTriSize); hipMalloc((void**)&d_BAQAY, rowActTriSize); hipMalloc((void**)&d_BAQAZ, rowActTriSize);
						hipMalloc((void**)&d_CBQBX, rowActTriSize); hipMalloc((void**)&d_CBQBY, rowActTriSize); hipMalloc((void**)&d_CBQBZ, rowActTriSize);
						hipMalloc((void**)&d_ACQCX, rowActTriSize); hipMalloc((void**)&d_ACQCY, rowActTriSize); hipMalloc((void**)&d_ACQCZ, rowActTriSize);

						vecCross << <width, numTris >> > (d_BAX, d_BAY, d_BAZ, d_QAX, d_QAY, d_QAZ, d_BAQAX, d_BAQAY, d_BAQAZ);
						vecCross << <width, numTris >> > (d_CBX, d_CBY, d_CBZ, d_QBX, d_QBY, d_QBZ, d_CBQBX, d_CBQBY, d_CBQBZ);
						vecCross << <width, numTris >> > (d_ACX, d_ACY, d_ACZ, d_QCX, d_QCY, d_QCZ, d_ACQCX, d_ACQCY, d_ACQCZ);

						hipDeviceSynchronize();

						hipFree(d_BAX); hipFree(d_BAY); hipFree(d_BAZ);
						hipFree(d_CBX); hipFree(d_CBY); hipFree(d_CBZ);
						hipFree(d_ACX); hipFree(d_ACY); hipFree(d_ACZ);

						hipFree(d_QAX); hipFree(d_QAY); hipFree(d_QAZ);
						hipFree(d_QBX); hipFree(d_QBY); hipFree(d_QBZ);
						hipFree(d_QCX); hipFree(d_QCY); hipFree(d_QCZ);

						int bvarsize = sizeof(bool) * numTris * width;
						hipMalloc((void**)&d_C1, bvarsize); hipMalloc((void**)&d_C2, bvarsize); hipMalloc((void**)&d_C3, bvarsize);

						hipMalloc((void**)&d_NX, size); hipMalloc((void**)&d_NY, size); hipMalloc((void**)&d_NZ, size);
						hipMemcpy(d_NX, NX, size, hipMemcpyHostToDevice); hipMemcpy(d_NY, NY, size, hipMemcpyHostToDevice); hipMemcpy(d_NZ, NZ, size, hipMemcpyHostToDevice);

						normDot << <width, numTris >> > (d_BAQAX, d_BAQAY, d_BAQAZ, d_NX, d_NY, d_NZ, d_C1);
						normDot << <width, numTris >> > (d_CBQBX, d_CBQBY, d_CBQBZ, d_NX, d_NY, d_NZ, d_C2);
						normDot << <width, numTris >> > (d_ACQCX, d_ACQCY, d_ACQCZ, d_NX, d_NY, d_NZ, d_C3);

						hipDeviceSynchronize();

						hipFree(d_BAQAX); hipFree(d_BAQAY); hipFree(d_BAQAZ);
						hipFree(d_CBQBX); hipFree(d_CBQBY); hipFree(d_CBQBZ);
						hipFree(d_ACQCX); hipFree(d_ACQCY); hipFree(d_ACQCZ);
						hipFree(d_NX); hipFree(d_NY); hipFree(d_NZ);


						bvarsize = sizeof(bool) * width;
						hipMalloc((void**)&d_fill, bvarsize);
						size = sizeof(int);
						hipMalloc((void**)&d_numTris, size);
						hipMemcpy(d_numTris, &numTris, size, hipMemcpyHostToDevice);

						checkSum << <width, 1 >> > (d_C1, d_C2, d_C3, d_numTris, d_fill);

						hipDeviceSynchronize();

						hipFree(d_C1); hipFree(d_C2); hipFree(d_C3); hipFree(d_numTris);

						size = sizeof(bool) * width;
						hipMemcpy(fills[d][h], d_fill, size, hipMemcpyDeviceToHost);
						hipFree(d_fill);
					}
				/*}
				else {
				for (int i = 0; i < width; i++) {
					fills[d][h][i] = false;
					}
				}*/
			}
		}
	}

	void voxelizeAngle(const std::vector<std::vector<int>> triVecs,
		const std::vector<std::vector<int>> norms,
		const int width, const int height, const int depth,
		const std::vector<int> minZTris,
		const std::vector<int> maxZTris,
		const std::vector<int> minYTris,
		const std::vector<int> maxYTris,
		bool*** fills) {


		int numTris = minZTris.size();
		int size;
		int * ax, * ay, * az,
			* bx, * by, * bz,
			* cx, * cy, * cz;
		int* NX, * NY, * NZ;
		int * d_ax, * d_ay, * d_az,
			* d_bx, * d_by, * d_bz,
			* d_cx, * d_cy, * d_cz;

		int * d_PAX, * d_PAY, * d_PAZ,
			* d_PBX, * d_PBY, * d_PBZ,
			* d_PCX, * d_PCY, * d_PCZ;

		float* d_AB, * d_BC, * d_CA;
		
		int* PLD; bool* d_plInt;
		int* d_PLA, * d_PLB, * d_PLC, * d_PLD;

		bool* d_intersects;
		bool* d_out;
		std::vector<int> activeTris;
		for (int d = 0; d < depth; d++) {
			for (int h = 0; h < height; h++) {
				//if (d == 9) {
					activeTris.clear();
					actTriFind(minZTris, maxZTris, minYTris, maxYTris, activeTris, h, d);
					int numTris = activeTris.size();
					size = sizeof(int) * numTris;
					ax = (int*)malloc(size); ay = (int*)malloc(size); az = (int*)malloc(size);
					bx = (int*)malloc(size); by = (int*)malloc(size); bz = (int*)malloc(size);
					cx = (int*)malloc(size); cy = (int*)malloc(size); cz = (int*)malloc(size);
					NX = (int*)malloc(size); NY = (int*)malloc(size); NZ = (int*)malloc(size);

					PLD = (int*)malloc(size);
					
					for (int i = 0; i < numTris; i++) {

						std::vector<int> actVecA = triVecs[(activeTris[i] * 3)];
						std::vector<int> actVecB = triVecs[(activeTris[i] * 3) + 1];
						std::vector<int> actVecC = triVecs[(activeTris[i] * 3) + 2];
						ax[i] = actVecA[0]; ay[i] = actVecA[1]; az[i] = actVecA[2];
						bx[i] = actVecB[0]; by[i] = actVecB[1]; bz[i] = actVecB[2];
						cx[i] = actVecC[0]; cy[i] = actVecC[1]; cz[i] = actVecC[2];
						std::vector<int> norm = norms[activeTris[i]];
						NX[i] = norm[0]; NY[i] = norm[1]; NZ[i] = norm[2];
						PLD[i] = norm[0] * ax[i] + norm[1] * ay[i] + norm[2] * az[i];
						norm.clear();
					}

					hipMalloc((void**)&d_ax, size); hipMalloc((void**)&d_ay, size); hipMalloc((void**)&d_az, size);
					hipMalloc((void**)&d_bx, size); hipMalloc((void**)&d_by, size); hipMalloc((void**)&d_bz, size);
					hipMalloc((void**)&d_cx, size); hipMalloc((void**)&d_cy, size); hipMalloc((void**)&d_cz, size);

					hipMemcpy(d_ax, ax, size, hipMemcpyHostToDevice); hipMemcpy(d_ay, ay, size, hipMemcpyHostToDevice); hipMemcpy(d_az, az, size, hipMemcpyHostToDevice);
					hipMemcpy(d_bx, bx, size, hipMemcpyHostToDevice); hipMemcpy(d_by, by, size, hipMemcpyHostToDevice); hipMemcpy(d_bz, bz, size, hipMemcpyHostToDevice);
					hipMemcpy(d_cx, cx, size, hipMemcpyHostToDevice); hipMemcpy(d_cy, cy, size, hipMemcpyHostToDevice); hipMemcpy(d_cz, cz, size, hipMemcpyHostToDevice);

					free(ax); free(ay); free(az);
					free(bx); free(by); free(bz);
					free(cx); free(cy); free(cz);

					hipMalloc((void**)&d_PLA, size); hipMalloc((void**)&d_PLB, size); hipMalloc((void**)&d_PLC, size); hipMalloc((void**)&d_PLD, size); 
					hipMemcpy(d_PLA, NX, size, hipMemcpyHostToDevice); hipMemcpy(d_PLB, NY, size, hipMemcpyHostToDevice); 
					hipMemcpy(d_PLC, NZ, size, hipMemcpyHostToDevice); hipMemcpy(d_PLD, PLD, size, hipMemcpyHostToDevice);

					size = sizeof(bool) * width * numTris;
					hipMalloc((void**)&d_plInt, size);

					int* d_PY, * d_PZ;
					size = sizeof(int);
					hipMalloc((void**)&d_PY, size); hipMalloc((void**)&d_PZ, size);
					hipMemcpy(d_PY, &h, size, hipMemcpyHostToDevice);
					hipMemcpy(d_PZ, &d, size, hipMemcpyHostToDevice);

					planeIntersect << <width, numTris >> > (d_PLA, d_PLB, d_PLC, d_PLD, d_PY, d_PZ, d_plInt);

					hipDeviceSynchronize();
					hipFree(d_PLA); hipFree(d_PLB); hipFree(d_PLC); hipFree(d_PLD);

					size = sizeof(int) * numTris * width;
					hipMalloc((void**)&d_PAX, size); hipMalloc((void**)&d_PAY, size); hipMalloc((void**)&d_PAZ, size);
					hipMalloc((void**)&d_PBX, size); hipMalloc((void**)&d_PBY, size); hipMalloc((void**)&d_PBZ, size);
					hipMalloc((void**)&d_PCX, size); hipMalloc((void**)&d_PCY, size); hipMalloc((void**)&d_PCZ, size);

					vecSubPointOri << <width, numTris >> > (d_PY, d_PZ, d_ax, d_ay, d_az, d_PAX, d_PAY, d_PAZ);
					vecSubPointOri << <width, numTris >> > (d_PY, d_PZ, d_bx, d_by, d_bz, d_PBX, d_PBY, d_PBZ);
					vecSubPointOri << <width, numTris >> > (d_PY, d_PZ, d_cx, d_cy, d_cz, d_PCX, d_PCY, d_PCZ);

					hipDeviceSynchronize();

					hipFree(d_PY); hipFree(d_PZ);

					hipFree(d_ax); hipFree(d_ay); hipFree(d_az);
					hipFree(d_bx); hipFree(d_by); hipFree(d_bz);
					hipFree(d_cx); hipFree(d_cy); hipFree(d_cz);

					size = sizeof(float) * numTris * width;
					hipMalloc((void**)&d_AB, size);
					hipMalloc((void**)&d_BC, size);
					hipMalloc((void**)&d_CA, size);

					angleFind << <width, numTris >> > (d_PAX, d_PAY, d_PAZ, d_PBX, d_PBY, d_PBZ, d_AB);
					angleFind << <width, numTris >> > (d_PBX, d_PBY, d_PBZ, d_PCX, d_PCY, d_PCZ, d_BC);
					angleFind << <width, numTris >> > (d_PCX, d_PCY, d_PCZ, d_PAX, d_PAY, d_PAZ, d_CA);

					hipDeviceSynchronize();

					hipFree(d_PAX); hipFree(d_PAY); hipFree(d_PAZ);
					hipFree(d_PBX); hipFree(d_PBY); hipFree(d_PBZ);
					hipFree(d_PCX); hipFree(d_PCY); hipFree(d_PCZ);

					size = sizeof(bool) * width * numTris;
					hipMalloc((void**)&d_intersects, size);
					//printf("height: %d\n", h);
					angleSum << <width, numTris >> > (d_AB, d_BC, d_CA, d_plInt, d_intersects);

					hipDeviceSynchronize();

					hipFree(d_AB); hipFree(d_BC); hipFree(d_CA);

					int* d_numTris;
					size = sizeof(int);
					hipMalloc((void**)&d_numTris, size);
					hipMemcpy(d_numTris, &numTris, size, hipMemcpyHostToDevice);

					size = sizeof(bool) * width;
					hipMalloc((void**)&d_out, size);

					intersectCount << <width, 1 >> > (d_numTris, d_intersects, d_out);
					hipFree(d_numTris); hipFree(d_intersects);

					hipMemcpy(fills[d][h], d_out, size, hipMemcpyDeviceToHost);
				//}
			}
		}
	}

	void rayVoxel(const std::vector<std::vector<int>> triVecs,
		const int width, const int height, const int depth,
		const std::vector<int> minZTris,
		const std::vector<int> maxZTris,
		const std::vector<int> minYTris,
		const std::vector<int> maxYTris,
		bool*** fills) {
		int size;
		int* ax, * ay, * az;
		int* d_ax, * d_ay, * d_az;

		int* NX, * NY, * NZ;
		int* d_NX, * d_NY, * d_NZ;

		int* ux, * uy, * uz,
			* vx, * vy, * vz;

		int* d_ux, * d_uy, * d_uz,
			* d_vx, * d_vy, * d_vz;

		int* d_wx, * d_wy, * d_wz;

		int* d_uu, * d_uv, * d_vv,
			* d_wu, * d_wv, * d_D;

		bool* d_intersects;
		bool* d_out;
		std::vector<int> activeTris;
		for (int d = 0; d < depth; d++) {
			for (int h = 0; h < height; h++) {
				if (d == 10) {
					if (h == 5) {
						activeTris.clear();
						actTriFind(minZTris, maxZTris, minYTris, maxYTris, activeTris, h, d);
						int numTris = activeTris.size();
						size = sizeof(int) * numTris;
						ax = (int*)malloc(size); ay = (int*)malloc(size); az = (int*)malloc(size);


						ux = (int*)malloc(size); uy = (int*)malloc(size); uz = (int*)malloc(size);
						vx = (int*)malloc(size); vy = (int*)malloc(size); vz = (int*)malloc(size);
						NX = (int*)malloc(size); NY = (int*)malloc(size); NZ = (int*)malloc(size);


						for (int i = 0; i < numTris; i++) {

							std::vector<int> actVecA = triVecs[(activeTris[i] * 3)];
							std::vector<int> actVecB = triVecs[(activeTris[i] * 3) + 1];
							std::vector<int> actVecC = triVecs[(activeTris[i] * 3) + 2];
							ax[i] = actVecA[0]; ay[i] = actVecA[1]; az[i] = actVecA[2];


							ux[i] = actVecB[0] - ax[i]; uy[i] = actVecB[1] - ay[i]; uz[i] = actVecB[2] - az[i];
							vx[i] = actVecC[0] - ax[i]; vy[i] = actVecC[1]; -ay[i]; vz[i] = actVecC[2] - az[i];
							NX[i] = (uy[i] * vz[i]) - (uz[i] * vy[i]);
							NY[i] = (uz[i] * vx[i]) - (ux[i] * vz[i]);
							NZ[i] = (ux[i] * vy[i]) - (uy[i] * vx[i]);
						}

						int* d_Z, * d_Y;
						size = sizeof(int); hipMalloc((void**)&d_Z, size); hipMalloc((void**)&d_Y, size);
						hipMemcpy(d_Z, &d, size, hipMemcpyHostToDevice); hipMemcpy(d_Y, &h, size, hipMemcpyHostToDevice);
						size = sizeof(int) * numTris * width; hipMalloc((void**)&d_wx, size); hipMalloc((void**)&d_wy, size); hipMalloc((void**)&d_wz, size);
						size = sizeof(int) * numTris; hipMalloc((void**)&d_ux, size); hipMalloc((void**)&d_uy, size); hipMalloc((void**)&d_uz, size);
						hipMalloc((void**)&d_vx, size); hipMalloc((void**)&d_vy, size); hipMalloc((void**)&d_vz, size);
						hipMemcpy(d_ux, ux, size, hipMemcpyHostToDevice); hipMemcpy(d_uy, uy, size, hipMemcpyHostToDevice); hipMemcpy(d_uz, uz, size, hipMemcpyHostToDevice);
						hipMemcpy(d_vx, vx, size, hipMemcpyHostToDevice); hipMemcpy(d_vy, vy, size, hipMemcpyHostToDevice); hipMemcpy(d_vz, vz, size, hipMemcpyHostToDevice);
						hipMalloc((void**)&d_ax, size); hipMalloc((void**)&d_ay, size); hipMalloc((void**)&d_az, size);
						hipMemcpy(d_ax, ax, size, hipMemcpyHostToDevice); hipMemcpy(d_ay, ay, size, hipMemcpyHostToDevice); hipMemcpy(d_az, az, size, hipMemcpyHostToDevice);
						free(ax); free(ay); free(az);
						vecSubPoint << <width, numTris >> > (d_Y, d_Z, d_ax, d_ay, d_az, d_wx, d_wy, d_wz);

						hipFree(d_ax); hipFree(d_ay); hipFree(d_az);
						hipFree(d_Y); hipFree(d_Z);

						hipMalloc((void**)&d_uu, size); hipMalloc((void**)&d_uv, size); hipMalloc((void**)&d_vv, size);
						normDotDouble << <numTris, 1 >> > (d_ux, d_uy, d_uz, d_ux, d_uy, d_uz, d_uu);
						normDotDouble << <numTris, 1 >> > (d_ux, d_uy, d_uz, d_vx, d_vy, d_vz, d_uv);
						normDotDouble << <numTris, 1 >> > (d_vx, d_vy, d_vz, d_vx, d_vy, d_vz, d_vv);

						hipMalloc((void**)&d_D, size);
						DCALC << <numTris, 1 >> > (d_uu, d_uv, d_vv, d_D);

						size = sizeof(int) * numTris * width;
						hipMalloc((void**)&d_wu, size); hipMalloc((void**)&d_wv, size);
						normDotW << <width, numTris >> > (d_wx, d_wy, d_wz, d_ux, d_uy, d_uz, d_wu);
						normDotW << <width, numTris >> > (d_wx, d_wy, d_wz, d_vx, d_vy, d_vz, d_wv);

						hipFree(d_wx); hipFree(d_wy); hipFree(d_wz);
						hipFree(d_ux); hipFree(d_uy); hipFree(d_uz);
						hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);

						size = sizeof(bool) * numTris * width;
						hipMalloc((void**)&d_intersects, size);

						paramTest << <width, numTris >> > (d_uu, d_uv, d_vv, d_wu, d_wv, d_D, d_intersects);

						hipFree(d_uu); hipFree(d_uv); hipFree(d_vv);
						hipFree(d_wu); hipFree(d_wv); hipFree(d_D);

						size = sizeof(bool) * width; hipMalloc((void**)&d_out, size);

						size = sizeof(int); int* d_numTris; hipMalloc((void**)&d_numTris, size); hipMemcpy(d_numTris, &numTris, size, hipMemcpyHostToDevice);

						intersectCount << <width, 1 >> > (d_numTris, d_intersects, d_out);
						hipFree(d_numTris); hipFree(d_intersects);
						hipMemcpy(fills[d][h], d_out, size, hipMemcpyDeviceToHost);
						hipFree(d_out);
					}
				}
			}
		}
	}
}