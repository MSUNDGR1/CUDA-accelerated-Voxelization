#include "hip/hip_runtime.h"
#include "voxelizer.cuh"


__global__ void vecSubDArr(int* firX, int* firY, int* firZ,
	int* secX, int* secY, int* secZ,
	int* outX, int* outY, int* outZ) {
	outX[blockIdx.x] = firX[blockIdx.x] - secX[blockIdx.x];
	outY[blockIdx.x] = firY[blockIdx.x] - secY[blockIdx.x];
	outZ[blockIdx.x] = firZ[blockIdx.x] - secZ[blockIdx.x];
}

__global__ void vecSubPoint(int* QY, int* QZ,
	int* VX, int* VY, int* VZ,
	int* outX, int* outY, int* outZ) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	outX[index] = blockIdx.x - VX[threadIdx.x];
	outY[index] = *QY - VY[threadIdx.x];
	outZ[index] = *QZ - VZ[threadIdx.x];
}

__global__ void vecCross(int* firX, int* firY, int* firZ,
	int* secX, int* secY, int* secZ,
	int* outX, int* outY, int* outZ) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	outX[index] = (firY[threadIdx.x] * secZ[index]) - (firZ[threadIdx.x] * secY[index]);
	outY[index] = (firZ[threadIdx.x] * secX[index]) - (firX[threadIdx.x] * secZ[index]);
	outZ[index] = (firX[threadIdx.x] * secY[index]) - (firY[threadIdx.x] * secX[index]);
}


__global__ void normDot(int* firX, int* firY, int* firZ,
	int* secX, int* secY, int* secZ,
	bool* check) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int sum = firX[index] * secX[threadIdx.x];
	sum += firY[index] * secY[threadIdx.x];
	sum += firZ[index] * secZ[threadIdx.x];
	if (sum >= 0) check[index] = true;
}

__global__ void checkSum(bool* c1, bool* c2, bool* c3,
	int* actTris, bool* fills) {
	bool checker = false;
	int index;
	int offset = blockIdx.x * (*actTris);
	for (int i = 0; i < (*actTris); i++) {
		index = offset + i;
		checker = (c1[index] | c2[index]) | c3[index];
		if (checker) fills[blockIdx.x] = true;
	}
}

void actTriFind(std::vector<int> minZTris,
	std::vector<int> maxZTris,
	std::vector<int> minYTris,
	std::vector<int> maxYTris,
	std::vector<int>& actTris,
	const int y, const int z) {
	int len = minZTris.size();
	bool checkZ;
	bool checkY;
	for (int i = 0; i < len; i++) {
		checkY = minYTris[i] <= y && maxYTris[i] >= y;
		checkZ = minZTris[i] <= z && maxZTris[i] >= z;
		if (checkY && checkZ) actTris.push_back(i);
	}
}

namespace voxel {
	void voxelize(
		const std::vector<std::vector<int>> triVecs,
		const std::vector<std::vector<int>> norms,
		const int width, const int height, const int depth,
		const std::vector<int> minZTris,
		const std::vector<int> maxZTris,
		const std::vector<int> minYTris,
		const std::vector<int> maxYTris,
		bool*** fills
	) {
		/*fills = new bool** [depth];
		for (int d = 0; d < depth; d++) {
			fills[d] = new bool* [height];
			for (int h = 0; h < height; h++) {
				fills[d][h] = new bool[width];
			}
		}*/
		int numTris = minZTris.size();
		int size;
		int* ax, * ay, * az,
			* bx, * by, * bz,
			* cx, * cy, * cz;

		int* NX, * NY, * NZ;

		int* d_ax, * d_ay, * d_az,
			* d_bx, * d_by, * d_bz,
			* d_cx, * d_cy, * d_cz;

		int* d_BAX, * d_BAY, * d_BAZ,
			* d_CBX, * d_CBY, * d_CBZ,
			* d_ACX, * d_ACY, * d_ACZ;

		int* d_QAX, * d_QAY, * d_QAZ,
			* d_QBX, * d_QBY, * d_QBZ,
			* d_QCX, * d_QCY, * d_QCZ;

		int* d_QY, * d_QZ;

		int* d_BAQAX, * d_BAQAY, * d_BAQAZ,
			* d_CBQBX, * d_CBQBY, * d_CBQBZ,
			* d_ACQCX, * d_ACQCY, * d_ACQCZ;

		int* d_NX, * d_NY, * d_NZ;
		bool* d_C1, * d_C2, * d_C3;
		bool* d_fill; int* d_numTris;
		std::vector<int> activeTris;
		for (int d = 0; d < depth; d++) {
			for (int h = 0; h < height; h++) {
				activeTris.clear();
				actTriFind(minZTris, maxZTris, minYTris, maxYTris, activeTris, h, d);
				int numTris = activeTris.size();
				size = sizeof(int) * numTris;
				ax = (int*)malloc(size); ay = (int*)malloc(size); az = (int*)malloc(size);
				bx = (int*)malloc(size); by = (int*)malloc(size); bz = (int*)malloc(size);
				cx = (int*)malloc(size); cy = (int*)malloc(size); cz = (int*)malloc(size);
				NX = (int*)malloc(size); NY = (int*)malloc(size); NZ = (int*)malloc(size);
				for (int i = 0; i < numTris; i++) {
					std::vector<int> actVecA = triVecs[(activeTris[i] * 3)];
					std::vector<int> actVecB = triVecs[(activeTris[i] * 3) + 1];
					std::vector<int> actVecC = triVecs[(activeTris[i] * 3) + 2];
					ax[i] = actVecA[0]; ay[i] = actVecA[1]; az[i] = actVecA[2];
					bx[i] = actVecB[0]; by[i] = actVecB[1]; bz[i] = actVecB[2];
					cx[i] = actVecC[0]; cy[i] = actVecC[1]; cz[i] = actVecC[2];
					NX[i] = norms[activeTris[i]][0]; NY[i] = norms[activeTris[i]][1]; NZ[i] = norms[activeTris[i]][2];
				}
				hipMalloc((void**)&d_ax, size); hipMalloc((void**)&d_ay, size); hipMalloc((void**)&d_az, size);
				hipMalloc((void**)&d_bx, size); hipMalloc((void**)&d_by, size); hipMalloc((void**)&d_bz, size);
				hipMalloc((void**)&d_cx, size); hipMalloc((void**)&d_cy, size); hipMalloc((void**)&d_cz, size);

				hipMalloc((void**)&d_BAX, size); hipMalloc((void**)&d_BAY, size); hipMalloc((void**)&d_BAZ, size);
				hipMalloc((void**)&d_CBX, size); hipMalloc((void**)&d_CBY, size); hipMalloc((void**)&d_CBZ, size);
				hipMalloc((void**)&d_ACX, size); hipMalloc((void**)&d_ACY, size); hipMalloc((void**)&d_ACZ, size);

				hipMemcpy(d_ax, ax, size, hipMemcpyHostToDevice); hipMemcpy(d_ay, ay, size, hipMemcpyHostToDevice); hipMemcpy(d_az, az, size, hipMemcpyHostToDevice);
				hipMemcpy(d_bx, bx, size, hipMemcpyHostToDevice); hipMemcpy(d_by, by, size, hipMemcpyHostToDevice); hipMemcpy(d_bz, bz, size, hipMemcpyHostToDevice);
				hipMemcpy(d_cx, cx, size, hipMemcpyHostToDevice); hipMemcpy(d_cy, cy, size, hipMemcpyHostToDevice); hipMemcpy(d_cz, cz, size, hipMemcpyHostToDevice);

				free(ax); free(ay); free(az);
				free(bx); free(by); free(bz);
				free(cx); free(cy); free(cz);

				vecSubDArr << <numTris, 1 >> > (d_bx, d_by, d_bz, d_ax, d_ay, d_az, d_BAX, d_BAY, d_BAZ);
				vecSubDArr << <numTris, 1 >> > (d_cx, d_cy, d_cz, d_bx, d_by, d_bz, d_CBX, d_CBY, d_CBZ);
				vecSubDArr << <numTris, 1 >> > (d_ax, d_ay, d_az, d_cx, d_cy, d_cz, d_ACX, d_ACY, d_ACZ);

				int rowActTriSize = sizeof(int) * numTris * width;

				hipMalloc((void**)&d_QAX, rowActTriSize); hipMalloc((void**)&d_QAY, rowActTriSize); hipMalloc((void**)&d_QAZ, rowActTriSize);
				hipMalloc((void**)&d_QBX, rowActTriSize); hipMalloc((void**)&d_QBY, rowActTriSize); hipMalloc((void**)&d_QBZ, rowActTriSize);
				hipMalloc((void**)&d_QCX, rowActTriSize); hipMalloc((void**)&d_QCY, rowActTriSize); hipMalloc((void**)&d_QCZ, rowActTriSize);

				int dvarsize = sizeof(int);
				hipMalloc((void**)&d_QY, dvarsize); hipMalloc((void**)&d_QZ, dvarsize);
				hipMemcpy(d_QY, &h, dvarsize, hipMemcpyHostToDevice); hipMemcpy(d_QZ, &d, dvarsize, hipMemcpyHostToDevice);

				vecSubPoint << <width, numTris >> > (d_QY, d_QZ, d_ax, d_ay, d_az, d_QAX, d_QAY, d_QAZ);
				vecSubPoint << <width, numTris >> > (d_QY, d_QZ, d_bx, d_by, d_bz, d_QBX, d_QBY, d_QBZ);
				vecSubPoint << <width, numTris >> > (d_QY, d_QZ, d_cx, d_cy, d_cz, d_QCX, d_QCY, d_QCZ);

				hipFree(d_QY); hipFree(d_QZ);

				hipFree(d_ax); hipFree(d_ay); hipFree(d_az);
				hipFree(d_bx); hipFree(d_by); hipFree(d_bz);
				hipFree(d_cx); hipFree(d_cy); hipFree(d_cz);

				hipMalloc((void**)&d_BAQAX, rowActTriSize); hipMalloc((void**)&d_BAQAY, rowActTriSize); hipMalloc((void**)&d_BAQAZ, rowActTriSize);
				hipMalloc((void**)&d_CBQBX, rowActTriSize); hipMalloc((void**)&d_CBQBY, rowActTriSize); hipMalloc((void**)&d_CBQBZ, rowActTriSize);
				hipMalloc((void**)&d_ACQCX, rowActTriSize); hipMalloc((void**)&d_ACQCY, rowActTriSize); hipMalloc((void**)&d_ACQCZ, rowActTriSize);

				vecCross << <width, numTris >> > (d_BAX, d_BAY, d_BAZ, d_QAX, d_QAY, d_QAZ, d_BAQAX, d_BAQAY, d_BAQAZ);
				vecCross << <width, numTris >> > (d_CBX, d_CBY, d_CBZ, d_QBX, d_QBY, d_QBZ, d_CBQBX, d_CBQBY, d_CBQBZ);
				vecCross << <width, numTris >> > (d_ACX, d_ACY, d_ACZ, d_QCX, d_QCY, d_QCZ, d_ACQCX, d_ACQCY, d_ACQCZ);

				hipFree(d_BAX); hipFree(d_BAY); hipFree(d_BAZ);
				hipFree(d_CBX); hipFree(d_CBY); hipFree(d_CBZ);
				hipFree(d_ACX); hipFree(d_ACY); hipFree(d_ACZ);

				hipFree(d_QAX); hipFree(d_QAY); hipFree(d_QAZ);
				hipFree(d_QBX); hipFree(d_QBY); hipFree(d_QBZ);
				hipFree(d_QCX); hipFree(d_QCY); hipFree(d_QCZ);

				int bvarsize = sizeof(bool) * numTris * width;
				hipMalloc((void**)&d_C1, bvarsize); hipMalloc((void**)&d_C2, bvarsize); hipMalloc((void**)&d_C3, bvarsize);

				hipMalloc((void**)&d_NX, size); hipMalloc((void**)&d_NY, size); hipMalloc((void**)&d_NZ, size);
				hipMemcpy(d_NX, NX, size, hipMemcpyHostToDevice); hipMemcpy(d_NY, NY, size, hipMemcpyHostToDevice); hipMemcpy(d_NZ, NZ, size, hipMemcpyHostToDevice);

				normDot << <width, numTris >> > (d_BAQAX, d_BAQAY, d_BAQAZ, d_NX, d_NY, d_NZ, d_C1);
				normDot << <width, numTris >> > (d_CBQBX, d_CBQBY, d_CBQBZ, d_NX, d_NY, d_NZ, d_C2);
				normDot << <width, numTris >> > (d_ACQCX, d_ACQCY, d_ACQCZ, d_NX, d_NY, d_NZ, d_C3);

				hipFree(d_BAQAX); hipFree(d_BAQAY); hipFree(d_BAQAZ);
				hipFree(d_CBQBX); hipFree(d_CBQBY); hipFree(d_CBQBZ);
				hipFree(d_ACQCX); hipFree(d_ACQCY); hipFree(d_ACQCZ);
				hipFree(d_NX); hipFree(d_NY); hipFree(d_NZ);


				bvarsize = sizeof(bool) * width;
				hipMalloc((void**)&d_fill, bvarsize);
				size = sizeof(int);
				hipMalloc((void**)&d_numTris, size);
				hipMemcpy(d_numTris, &numTris, size, hipMemcpyHostToDevice);

				checkSum << <width, 1 >> > (d_C1, d_C2, d_C3, d_numTris, d_fill);

				hipFree(d_C1); hipFree(d_C2); hipFree(d_C3); hipFree(d_numTris);

				size = sizeof(bool) * width;
				hipMemcpy(fills[d][h], d_fill, size, hipMemcpyDeviceToHost);
				hipFree(d_fill);
			}
		}
	}
}